#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include "real.h"

/*
 * ======================= advect1 ====================
 * Integrate forward (advection only) by one time step.
 * ATMS 502 / CSE 566, Spring 2016
 *
 * Arguments:
 *
 *	q1	real array	values at current step
 *	q2	real array	values at next step
 *	uv	real array	true speed of wave
 *	dx	real		grid spacing
 *	dt	real		time step
 *	iW,iE	integers	indices bounding array data
 *	advection_type
 *              char 		if 'L', linear advection;
 *				otherwise, nonlinear
 */

void advec1(real *__restrict__  q2,const real *__restrict__  q1,real *uv,real dx,real dt,int iW,int iE, int nxydim, char advection_type)
{
    real courant;
    //real *flux; 
    
	
    switch ( advection_type ) {
    case 'l':
        for (int i=iW; i<=iE; ++i) {
            courant = uv[i] * dt / dx;
            q2[i] = q1[i] - 0.5 * courant * ((q1[i+1] - q1[i-1]) - courant*(q1[i+1] -2.0*q1[i] + q1[i-1]));
        } // end for //
        break;
    case 'c':
        for (int i=iW, k=0; i<=iE; ++i, ++k) {
            courant = uv[i] * dt / dx;
            q2[i] = q1[i] + 
            courant*                                        (  (q1[i-3]-q1[i+3]) -  9*(q1[i-2]-q1[i+2]) +  45*(q1[i-1]-q1[i+1])            ) /  60.0 +
            courant*courant*                                (2*(q1[i-3]+q1[i+3]) - 27*(q1[i-2]+q1[i+2]) + 270*(q1[i-1]+q1[i+1]) - 490*q1[i]) / 360.0 +
            courant*courant*courant*                        ( -(q1[i-3]-q1[i+3]) +  8*(q1[i-2]-q1[i+2]) -  13*(q1[i-1]-q1[i+1])            ) /  48.0 +
            courant*courant*courant*courant*                ( -(q1[i-3]+q1[i+3]) + 12*(q1[i-2]+q1[i+2]) -  39*(q1[i-1]+q1[i+1]) +  56*q1[i]) / 144.0 +
            courant*courant*courant*courant*courant*        (  (q1[i-3]-q1[i+3]) -  4*(q1[i-2]-q1[i+2]) +   5*(q1[i-1]-q1[i+1])            ) / 240.0 +
            courant*courant*courant*courant*courant*courant*(  (q1[i-3]+q1[i+3]) -  6*(q1[i-2]+q1[i+2]) +  15*(q1[i-1]+q1[i+1]) -  20*q1[i]) / 720.0;
        } // end for //
        break;
    case 't':
        for (int i=iW; i<=iE; ++i) {
            courant = uv[i] * dt / dx;
            q2[i] = q1[i] + 0.5*courant*( (q1[i-1] - q1[i+1]) + courant*(q1[i+1] -2.0*q1[i] + q1[i-1]));
            if (courant >= 0) {
                q2[i] -= ( (1+courant)      *courant*(courant-1)/6.0*(q1[i+1] - 3*q1[i] + 3*q1[i-1] - q1[i-2])  );         
            } else {
                q2[i] -= ( (1+fabs(courant))*courant*(courant+1)/6.0*(q1[i-1] - 3*q1[i] + 3*q1[i+1] - q1[i+2])  );    
            } // end if //         
        } // end for //
        break;
    case 'p':
    /*
        {
        real *__restrict__   flux = (real *) malloc( nxydim*sizeof(real) );
        for (int i=iW; i<=iE+1; ++i) {
            real r = fabs(uv[i]*dt/dx);
            if (uv[i] >=0.0) {
                flux[i]= r*(q1[i-1] + 0.25*(1-r)*(q1[i]-q1[i-2]));
            } else {
                flux[i]= r*(-q1[i]  + 0.25*(1-r)*(q1[i+1] - q1[i-1]));	    
            }
        } // end for //
        for (int i=iW; i<=iE; ++i) {
            q2[i] = q1[i] - (flux[i+1] - flux[i]) + dt/dx*q1[i]*(uv[i+1] - uv[i]);
        } // end for 
        free(flux);
        }
        */
        real flux1,flux2,r;
        r = fabs(uv[iW]*dt/dx);

        if (uv[iW] >=0.0) {
            flux1= r*(q1[iW-1] + 0.25*(1-r)*(q1[iW]-q1[iW-2]));
        } else {
            flux1= r*(-q1[iW]  + 0.25*(1-r)*(q1[iW+1] - q1[iW-1]));	    
        } // end if 
        
        for (int i=iW; i<=iE; ++i) {
            r = fabs(uv[i+1]*dt/dx);
            if (uv[i+1] >=0.0) {
                flux2= r*(q1[i] + 0.25*(1-r)*(q1[i+1]-q1[i-1]));
            } else {
                flux2= r*(-q1[i+1]  + 0.25*(1-r)*(q1[i+2] - q1[i]));	    
            } // end if //
            q2[i] = q1[i] - (flux2 - flux1) + dt/dx*q1[i]*(uv[i+1] - uv[i]);
            flux1=flux2;
        } // end for 
        break;
    case 'u':
        for (int i=iW; i<=iE; ++i) {
            courant = uv[i] * dt / dx;
            if (courant >= 0.0) {
                q2[i] = q1[i] - courant*(q1[i] - q1[i-1]);
            } else {
                q2[i] = q1[i] - courant*(q1[i+1] - q1[i]);
            } // end if //
        } // end for //
        break;
    case 'n':
        for (int i=iW, k=0; i<=iE; ++i,++k) {
            courant = q1[k] * dt / dx;
            q2[i] = q1[i] - 0.5*courant * ((q1[i+1] - q1[i-1]) - courant*(q1[i+1] -2.0*q1[i] + q1[i-1]));
        } // end for //
        break;
    default:
        cout << "Integrate: Error, unrecognized advection type " << advection_type << endl;
        exit(-1);
        break;
    } // end switch //

} // end of advec1() //
